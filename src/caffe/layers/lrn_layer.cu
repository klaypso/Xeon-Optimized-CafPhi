#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void LRNFillScale(const int nthreads, const Dtype* in,
    const int num, const int channels, const int height,
    const int width, const int size, const Dtype alpha_over_size,
    const Dtype k, Dtype* scale) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height;
    int offset = (n * channels * height + h) * width + w;
    int step = height * width;
    in += offset;
    scale += offset;
    int head = 0;
    int pre_pad = (size - 1) / 2;
    int post_pad = size - pre_pad - 1;
    Dtype accum_scale = 0;
    // fill the scale at [n, :, h, w]
    // accumulate values
    while (head < post_pad && head < channels) {
      accum_scale += in[head * step] * in[head * step];
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in[head * step] * in[head * step];
      if (head - size >= 0) {
        accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      }
      scale[(head - post_pad) * step] = k + accum_scale * alpha_over_size;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      if (head - size >= 0) {
        accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      }
      scale[(head - post_pad) * step] = k + accum_scale * alpha_over_size;
      ++head;
    }
  }
}


template <typename Dtype>
void LRNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  swit