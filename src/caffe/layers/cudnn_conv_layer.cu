#include "hip/hip_runtime.h"
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    size_t workspace_limit_bytes = this->kernel_h_ *
                                   this->kernel_w_ *
                                   this->channels_ *
                                   sizeof(int) + 1;

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      hipdnnConvolutionFwdAlgo_t algo;

      // pick the convolution algorithm
      // TODO(shelhamer) this should be done during reshape
      // TODO(shelhamer) the choice of automatic or manual algorithm picking
      // should be exposed in proto
      CUDNN_CHECK(hipdnnGetConvolutionForwardAlgorithm(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_limit_bytes,  // memoryLimitInBytes,
        &algo));

      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes_temp = 0;

      CUDNN_CHECK(hipdnnGetConvolutionForwardWorkspaceSize(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        algo,
        &workspaceSizeInBytes_temp));

      if (workspaceSizeInBytes_temp > workspaceSizeInBytes) {
        workspaceSizeInBytes = workspaceSizeInBytes_temp;
        // free the existing workspace and allocate a new (larger) one
        hipFree(this->workspace);
        hipError_t err = hipMalloc(&(this->workspace), workspaceSizeInBytes);
        if (err != hipSuccess) {
  